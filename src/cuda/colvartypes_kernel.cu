#include "hip/hip_runtime.h"
#include "colvar_gpu_support.h"
#include "colvartypes_kernel.h"
#include "colvartypes.h"

#if defined(COLVARS_CUDA)
#include <cub/block/block_reduce.cuh>
#endif

// TODO: HIP CUB

namespace colvars_gpu {
#if defined(COLVARS_CUDA) || defined(COVLARS_HIP)

template <int BLOCK_SIZE>
__global__ void build_overlapping_matrix_kernel(
  const cvm::real* __restrict pos1_x,
  const cvm::real* __restrict pos1_y,
  const cvm::real* __restrict pos1_z,
  const cvm::real* __restrict pos2_x,
  const cvm::real* __restrict pos2_y,
  const cvm::real* __restrict pos2_z,
  cvm::real* __restrict S,
  cvm::real* __restrict S_eigvec,
  cvm::rmatrix* __restrict h_C,
  unsigned int* __restrict tbcount,
  int num_atoms) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ bool isLastBlockDone;
  if (threadIdx.x == 0) {
    isLastBlockDone = false;
  }
  __syncthreads();
  cvm::rmatrix C;
  C.reset();
  if (i < num_atoms) {
    C.xx = pos1_x[i] * pos2_x[i];
    C.xy = pos1_x[i] * pos2_y[i];
    C.xz = pos1_x[i] * pos2_z[i];
    C.yx = pos1_y[i] * pos2_x[i];
    C.yy = pos1_y[i] * pos2_y[i];
    C.yz = pos1_y[i] * pos2_z[i];
    C.zx = pos1_z[i] * pos2_x[i];
    C.zy = pos1_z[i] * pos2_y[i];
    C.zz = pos1_z[i] * pos2_z[i];
  }
  __syncthreads();
  typedef hipcub::BlockReduce<double, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  C.xx = BlockReduce(temp_storage).Sum(C.xx); __syncthreads();
  C.xy = BlockReduce(temp_storage).Sum(C.xy); __syncthreads();
  C.xz = BlockReduce(temp_storage).Sum(C.xz); __syncthreads();
  C.yx = BlockReduce(temp_storage).Sum(C.yx); __syncthreads();
  C.yy = BlockReduce(temp_storage).Sum(C.yy); __syncthreads();
  C.yz = BlockReduce(temp_storage).Sum(C.yz); __syncthreads();
  C.zx = BlockReduce(temp_storage).Sum(C.zx); __syncthreads();
  C.zy = BlockReduce(temp_storage).Sum(C.zy); __syncthreads();
  C.zz = BlockReduce(temp_storage).Sum(C.zz); __syncthreads();
  if (threadIdx.x == 0) {
    // S is 4 x 4 so I can use it as a temporary buffer
    atomicAdd(&(S[0]), C.xx);
    atomicAdd(&(S[1]), C.xy);
    atomicAdd(&(S[2]), C.xz);
    atomicAdd(&(S[3]), C.yx);
    atomicAdd(&(S[4]), C.yy);
    atomicAdd(&(S[5]), C.yz);
    atomicAdd(&(S[6]), C.zx);
    atomicAdd(&(S[7]), C.zy);
    atomicAdd(&(S[8]), C.zz);
    __threadfence();
    unsigned int value = atomicInc(tbcount, gridDim.x);
    isLastBlockDone = (value == (gridDim.x - 1));
  }
  __syncthreads();
  if (isLastBlockDone) {
    if (threadIdx.x == 0) {
      C.xx = S[0];
      C.xy = S[1];
      C.xz = S[2];
      C.yx = S[3];
      C.yy = S[4];
      C.yz = S[5];
      C.zx = S[6];
      C.zy = S[7];
      C.zz = S[8];
      // Now we can use the first thread of the last block to set S
      S[0*4+0] =   C.xx + C.yy + C.zz;
      S[1*4+0] =   C.yz - C.zy;
      S[0*4+1] =   C.yz - C.zy;
      S[2*4+0] = - C.xz + C.zx ;
      S[0*4+2] = - C.xz + C.zx ;
      S[3*4+0] =   C.xy - C.yx;
      S[0*4+3] =   C.xy - C.yx;
      S[1*4+1] =   C.xx - C.yy - C.zz;
      S[2*4+1] =   C.xy + C.yx;
      S[1*4+2] =   C.xy + C.yx;
      S[3*4+1] =   C.xz + C.zx;
      S[1*4+3] =   C.xz + C.zx;
      S[2*4+2] = - C.xx + C.yy - C.zz;
      S[3*4+2] =   C.yz + C.zy;
      S[2*4+3] =   C.yz + C.zy;
      S[3*4+3] = - C.xx - C.yy + C.zz;
      S_eigvec[0*4+0] =   C.xx + C.yy + C.zz;
      S_eigvec[1*4+0] =   C.yz - C.zy;
      S_eigvec[0*4+1] =   C.yz - C.zy;
      S_eigvec[2*4+0] = - C.xz + C.zx ;
      S_eigvec[0*4+2] = - C.xz + C.zx ;
      S_eigvec[3*4+0] =   C.xy - C.yx;
      S_eigvec[0*4+3] =   C.xy - C.yx;
      S_eigvec[1*4+1] =   C.xx - C.yy - C.zz;
      S_eigvec[2*4+1] =   C.xy + C.yx;
      S_eigvec[1*4+2] =   C.xy + C.yx;
      S_eigvec[3*4+1] =   C.xz + C.zx;
      S_eigvec[1*4+3] =   C.xz + C.zx;
      S_eigvec[2*4+2] = - C.xx + C.yy - C.zz;
      S_eigvec[3*4+2] =   C.yz + C.zy;
      S_eigvec[2*4+3] =   C.yz + C.zy;
      S_eigvec[3*4+3] = - C.xx - C.yy + C.zz;
      // Save the data to host memory
      memcpy(h_C, &C, sizeof(cvm::rmatrix));
    }
  }
}

int build_overlapping_matrix(
  const cvm::real* pos1_x,
  const cvm::real* pos1_y,
  const cvm::real* pos1_z,
  const cvm::real* pos2_x,
  const cvm::real* pos2_y,
  const cvm::real* pos2_z,
  cvm::real* S,
  cvm::real* S_eigvec,
  cvm::rmatrix* h_C,
  unsigned int* tbcount,
  int num_atoms,
  hipGraphNode_t& node,
  hipGraph_t& graph,
  const std::vector<hipGraphNode_t>& dependencies) {
  // if (num_atoms == 0) return;
  const int block_size = default_block_size;
  const int num_blocks = (num_atoms + block_size - 1) / block_size;
  void* args[] = {
    &pos1_x, &pos1_y, &pos1_z,
    &pos2_x, &pos2_y, &pos2_z,
    &S, &S_eigvec, &h_C, &tbcount, &num_atoms};
  hipKernelNodeParams kernelNodeParams = {0};
  kernelNodeParams.func           =
    (void*)build_overlapping_matrix_kernel<block_size>;
  kernelNodeParams.gridDim        = dim3(num_blocks, 1, 1);
  kernelNodeParams.blockDim       = dim3(block_size, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.kernelParams   = args;
  kernelNodeParams.extra          = NULL;
  if (cvm::debug()) {
    cvm::log("Add " + cvm::to_str(__func__) + " node.\n");
  }
  return checkGPUError(hipGraphAddKernelNode(
    &node, graph, dependencies.data(),
    dependencies.size(), &kernelNodeParams));
}

__global__ void eigvec_get_q_kernel(
  const cvm::real* S_eigvec,
  cvm::quaternion* q,
  bool monitor_crossings,
  cvm::real crossing_threshold,
  cvm::quaternion* q_old,
  int* discontinuous_rotation) {
  if (threadIdx.x == 0) {
    const int max_eigenvalue_index = 0;
    q->q0 = S_eigvec[max_eigenvalue_index*4+0];
    q->q1 = S_eigvec[max_eigenvalue_index*4+1];
    q->q2 = S_eigvec[max_eigenvalue_index*4+2];
    q->q3 = S_eigvec[max_eigenvalue_index*4+3];
    if (monitor_crossings) {
      if (q_old->norm2() > 0) {
        q->match(*q_old);
        if (q_old->inner(*q) < (1.0 - crossing_threshold)) {
          atomicAdd(discontinuous_rotation, 1);
        }
      }
    }
  }
}

int eigvec_get_q(
  const cvm::real* S_eigvec,
  cvm::quaternion* q,
  bool monitor_crossings,
  cvm::real crossing_threshold,
  cvm::quaternion* q_old,
  int* discontinuous_rotation,
  hipGraphNode_t& node,
  hipGraph_t& graph,
  const std::vector<hipGraphNode_t>& dependencies) {
  // const int block_size = default_block_size;
  void* args[] = {
    &S_eigvec, &q, &monitor_crossings,
    &crossing_threshold, &q_old,
    &discontinuous_rotation};
  hipKernelNodeParams kernelNodeParams = {0};
  kernelNodeParams.func           = (void*)eigvec_get_q_kernel;
  kernelNodeParams.gridDim        = dim3(1, 1, 1);
  kernelNodeParams.blockDim       = dim3(1, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.kernelParams   = args;
  kernelNodeParams.extra          = NULL;
  if (cvm::debug()) {
    cvm::log("Add " + cvm::to_str(__func__) + " node.\n");
  }
  return checkGPUError(hipGraphAddKernelNode(
    &node, graph, dependencies.data(),
    dependencies.size(), &kernelNodeParams));
}

#define JACOBI_MAX_ITERATION 50
#define JACOBI_TOLERANCE 1e-16
__inline__ __device__ void apply_jacobi(
  // const double* __restrict old_A,
  double* __restrict A,
  int p, int q, double c, double s) {
  const double c2 = c*c;
  const double s2 = s*s;
  const double cs = c*s;
  #pragma unroll
  for (int i = 0; i < 4; ++i) {
    const double oip = A[i*4+p];
    const double oiq = A[i*4+q];
    if (i != p && i != q) {
      A[i*4+p] = c * oip - s * oiq;
      A[p*4+i] = A[i*4+p];
      A[i*4+q] = c * oiq + s * oip;
      A[q*4+i] = A[i*4+q];
    }
  }
  const double opp = A[p*4+p];
  const double oqq = A[q*4+q];
  const double opq = A[p*4+q];
  A[p*4+p] = c2 * opp + s2 * oqq - 2.0 * cs * opq;
  A[q*4+q] = s2 * opp + c2 * oqq + 2.0 * cs * opq;
  A[p*4+q] = 0;
  A[q*4+p] = 0;
}

__inline__ __device__ void multiply_jacobi(
  double* __restrict V, int p, int q, double c, double s) {
  #pragma unroll
  for (int i = 0; i < 4; ++i) {
    const double oip = V[i*4+p];
    const double oiq = V[i*4+q];
    V[i*4+p] = c * oip - s * oiq;
    V[i*4+q] = s * oip + c * oiq;
  }
}

__inline__ __device__ void compute_c_s(
  double a_pq, double a_pp, double a_qq, double& c, double& s) {
  const double theta = 0.5 * (a_qq - a_pp) / a_pq;
  const double t = 1 / (sqrt(theta * theta + 1.0) + fabs(theta));
  // const double t = sqrt(theta * theta + 1.0) - fabs(theta);
  c = rsqrt(t * t + 1.0);
  s = theta < 0 ? -t * c : t * c;
  // const double phi = 0.5 * atan2(2 * a_pq, a_qq - a_pp);
  // sincos(phi, &s, &c);
}

// Use exactly 2 threads
__global__ void jacobi_4x4_kernel(
  double* A_in, double* eigvals, int* max_reached) {
  __shared__ double A[4*4];
  __shared__ double V[4*4];
  const int idx = threadIdx.x;
  if (max_reached && idx == 0) {
    max_reached[0] = 0;
    __threadfence();
  }
  if (idx == 0) {
    memset(V, 0, sizeof(double)*4*4);
    V[0*4+0] = 1;
    V[1*4+1] = 1;
    V[2*4+2] = 1;
    V[3*4+3] = 1;
    A[0] = A_in[0];
    A[1] = A_in[1];
    A[2] = A_in[2];
    A[3] = A_in[3];
    A[4] = A_in[4];
    A[5] = A_in[5];
    A[6] = A_in[6];
    A[7] = A_in[7];
    A[8] = A_in[8];
    A[9] = A_in[9];
    A[10] = A_in[10];
    A[11] = A_in[11];
    A[12] = A_in[12];
    A[13] = A_in[13];
    A[14] = A_in[14];
    A[15] = A_in[15];
  }
  __syncthreads();
  const int p_ids[] = {0, 2, 0, 1, 0, 1};
  const int q_ids[] = {1, 3, 2, 3, 3, 2};
  double off_diag_sum =
    A[0*4+1]*A[0*4+1]+A[0*4+2]*A[0*4+2]+A[0*4+3]*A[0*4+3]+
    A[1*4+2]*A[1*4+2]+A[1*4+3]*A[1*4+3]+
    A[2*4+3]*A[2*4+3];
  int iteration = 0;
  while (off_diag_sum > JACOBI_TOLERANCE) {
    double c = 0, s = 0;
    bool rotate = false;
    int p = p_ids[idx];
    int q = q_ids[idx];
    double a_pq = A[p*4+q];
    if (fabs(a_pq) > 0) {
      rotate = true;
      const double a_pp = A[p*4+p];
      const double a_qq = A[q*4+q];
      compute_c_s(a_pq, a_pp, a_qq, c, s);
    }
    __syncwarp();
    if (idx == 0 && rotate) {
      apply_jacobi(A, 0, 1, c, s);
      multiply_jacobi(V, 0, 1, c, s);
    }
    __syncwarp();
    if (idx == 1 && rotate) {
      apply_jacobi(A, 2, 3, c, s);
      multiply_jacobi(V, 2, 3, c, s);
    }
    __syncwarp();
    rotate = false;
    p = p_ids[idx+2];
    q = q_ids[idx+2];
    a_pq = A[p*4+q];
    if (fabs(a_pq) > 0) {
      rotate = true;
      const double a_pp = A[p*4+p];
      const double a_qq = A[q*4+q];
      compute_c_s(a_pq, a_pp, a_qq, c, s);
    }
    __syncwarp();
    if (idx == 0 && rotate) {
      apply_jacobi(A, 0, 2, c, s);
      multiply_jacobi(V, 0, 2, c, s);
    }
    __syncwarp();
    if (idx == 1 && rotate) {
      apply_jacobi(A, 1, 3, c, s);
      multiply_jacobi(V, 1, 3, c, s);
    }
    __syncwarp();
    rotate = false;
    p = p_ids[idx+4];
    q = q_ids[idx+4];
    a_pq = A[p*4+q];
    if (fabs(a_pq) > 0) {
      rotate = true;
      const double a_pp = A[p*4+p];
      const double a_qq = A[q*4+q];
      compute_c_s(a_pq, a_pp, a_qq, c, s);
    }
    __syncwarp();
    if (idx == 0 && rotate) {
      apply_jacobi(A, 0, 3, c, s);
      multiply_jacobi(V, 0, 3, c, s);
    }
    __syncwarp();
    if (idx == 1 && rotate) {
      apply_jacobi(A, 1, 2, c, s);
      multiply_jacobi(V, 1, 2, c, s);
    }
    __syncwarp();
    off_diag_sum =
      A[0*4+1]*A[0*4+1]+A[0*4+2]*A[0*4+2]+A[0*4+3]*A[0*4+3]+
      A[1*4+2]*A[1*4+2]+A[1*4+3]*A[1*4+3]+
      A[2*4+3]*A[2*4+3];
    // Check the number of iterations
    ++iteration;
    if (iteration > JACOBI_MAX_ITERATION) {
      if (idx == 0 && max_reached) atomicAdd(max_reached, 1);
      break;
    }
  }
  // Sort
  double p;
  if (idx == 0) {
    int k;
    #pragma unroll
    for (int i0 = 0; i0 < 4; ++i0) {
      k = i0;
      p = A[i0*4+i0];
      for (int j0 = i0 + 1; j0 < 4; ++j0) {
        // Using ">=" is much slower
        if (A[j0*4+j0] > p) {
          k = j0;
          p = A[j0*4+j0];
        }
      }
      if (k != i0) {
        A[k*4+k] = A[i0*4+i0];
        A[i0*4+i0] = p;
        for (int j0 = 0; j0 < 4; ++j0) {
          p = V[j0*4+i0];
          V[j0*4+i0] = V[j0*4+k];
          V[j0*4+k] = p;
        }
      }
    }
    A_in[0] = V[0];
    A_in[1] = V[4];
    A_in[2] = V[8];
    A_in[3] = V[12];
    A_in[4] = V[1];
    A_in[5] = V[5];
    A_in[6] = V[9];
    A_in[7] = V[13];
    A_in[8] = V[2];
    A_in[9] = V[6];
    A_in[10] = V[10];
    A_in[11] = V[14];
    A_in[12] = V[3];
    A_in[13] = V[7];
    A_in[14] = V[11];
    A_in[15] = V[15];
    eigvals[0] = A[0*4+0];
    eigvals[1] = A[1*4+1];
    eigvals[2] = A[2*4+2];
    eigvals[3] = A[3*4+3];
  }
}
#undef JACOBI_MAX_ITERATION
#undef JACOBI_TOLERANCE

int jacobi_4x4(
  double* S_eigvec,
  double* S_eigval,
  int* max_reached,
  hipGraphNode_t& node,
  hipGraph_t& graph,
  const std::vector<hipGraphNode_t>& dependencies) {
  void* args[] = {
    &S_eigvec, &S_eigval, &max_reached};
  hipKernelNodeParams kernelNodeParams = {0};
  kernelNodeParams.func           = (void*)jacobi_4x4_kernel;
  kernelNodeParams.gridDim        = dim3(1, 1, 1);
  kernelNodeParams.blockDim       = dim3(2, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.kernelParams   = args;
  kernelNodeParams.extra          = NULL;
  if (cvm::debug()) {
    cvm::log("Add " + cvm::to_str(__func__) + " node.\n");
  }
  return checkGPUError(hipGraphAddKernelNode(
    &node, graph, dependencies.data(),
    dependencies.size(), &kernelNodeParams));
}

#elif defined(COLVARS_SYCL)
#endif // defined(COLVARS_CUDA) || defined(COVLARS_HIP)
}
